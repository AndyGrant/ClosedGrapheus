#include "hip/hip_runtime.h"

#include "relu.h"

#include <iostream>

// clang-format off
void operations::relu_host(
    const float* A,
          float* B,
    size_t m,
    size_t n,
    size_t lda,
    size_t ldb){
    // clang-format on

    for (int x = 0; x < n; x++){
        for(int y = 0; y < m; y++){
            int ida = MATRIX_INDEX(lda, y, x);
            int idb = MATRIX_INDEX(ldb, y, x);

            B[idb] = std::max(0.0f, A[ida]);
        }
    }

}
