#include "hip/hip_runtime.h"

#include "relu_bp.h"

// clang-format off
__global__ void operations::relu_bp_kernel(
    const float* __restrict__ A,
          float* __restrict__ A_grd,
    const float* __restrict__ B,
    const float* __restrict__ B_grd,
    size_t m,
    size_t n,
    size_t lda,
    size_t ldb){
    // clang-format on

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx >= n || idy >= m)
        return;

    int ida = MATRIX_INDEX(lda, idy, idx);
    int idb = MATRIX_INDEX(ldb, idy, idx);

    if (B[idb] > 0) {
        A_grd[ida] = B_grd[idb];
    } else {
        A_grd[ida] = 0;
    }
}

// a fast version which assumes leading dimension of both matrices is m
// to be precise this assumes that the matrix itself is not a submatrix
// clang-format off
__global__ void operations::relu_bp_kernel_fast(
    const float* __restrict__ A,
          float* __restrict__ A_grd,
    const float* __restrict__ B,
    const float* __restrict__ B_grd,
    size_t size){
    // clang-format on

    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx > size) return;

    if (B[idx] > 0) {
        A_grd[idx] = B_grd[idx];
    } else {
        A_grd[idx] = 0;
    }
}